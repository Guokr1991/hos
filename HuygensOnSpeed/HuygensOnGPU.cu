#include "hip/hip_runtime.h"
#include "HuygensOnGPU.h"

#include "HuygensKernel.cu"

hipComplex* HuygensOnGPU::calcFieldResponse(
	ObservationArea* obsArea,
	std::vector<ISource<float>*> &src,
	const float timestampObs)
{
	return 0;
}

void HuygensOnGPU::calcFieldResponse(hipComplex* d_res,
									 const unsigned int nObs, const float* coordObs,	// Observation # and coordiantes
									 const unsigned int nSrc, const float* coordSrc,	// Source #, coordinates,
									 const float* fSrc, const float* apodSrc,	// frequencies, apodization
									 const float* steerFocusDelaySrc,			// and steer-focus delays
									 const float* srcTimeStamp,					// time stamp telling when source starts to fire
									 const unsigned int* srcPulseLength,				// pulse length 0 == Inf
									 const float timestampObs,					// Current timestamp for this observation
									 const float refTime,						// Reference time for calculating attenuation 	
									 const float c0,
									 const bool resultOnGPU)					// true if d_res is on the GPU
{

#ifdef USE_CONST_MEMORY // Give this def to preprocessor for const memory. Use constant memory for devices of cc < 2.0  
	if (nSrc > CONST_MEM_SIZE) {
		fprintf(stderr, "Maximum number of sources is restricted to %d when constant memory is used\n", CONST_MEM_SIZE);
	} 
	else
	{
#endif

		//cuUtilSetDevice(); // select device (max sm count)

		// copy points etc which is not in device memory to the device
		float* d_coordObs = NULL;
		unsigned int memSize = sizeof(float) * nObs * 3;
		cuUtilsSafeCall( hipMalloc<float>(&d_coordObs, memSize) );
		cuUtilsSafeCall( hipMemcpy(d_coordObs, coordObs, memSize, hipMemcpyHostToDevice) );

		float* d_coordSrc = NULL;
		memSize = sizeof(float) * nSrc * 3;
		cuUtilsSafeCall( hipMalloc<float>(&d_coordSrc, memSize) );
		cuUtilsSafeCall( hipMemcpy(d_coordSrc, coordSrc, memSize, hipMemcpyHostToDevice) );

		float* d_fSrc = NULL;
		memSize = sizeof(float) * nSrc;

#ifdef USE_CONST_MEMORY
		cuUtilsSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_fSrc), fSrc, memSize) );
#else
		cuUtilsSafeCall( hipMalloc<float>(&d_fSrc, memSize) );
		cuUtilsSafeCall( hipMemcpy(d_fSrc, fSrc, memSize, hipMemcpyHostToDevice) );
#endif	

		float* d_apodSrc = NULL;
#ifdef USE_CONST_MEMORY
		cuUtilsSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_apodSrc), apodSrc, memSize) );
#else
		cuUtilsSafeCall( hipMalloc<float>(&d_apodSrc, memSize) );
		cuUtilsSafeCall( hipMemcpy(d_apodSrc, apodSrc, memSize, hipMemcpyHostToDevice) );
#endif
		

		float* d_steerFocusDelaySrc = NULL;
#ifdef USE_CONST_MEMORY
		cuUtilsSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_steerFocusDelaySrc), steerFocusDelaySrc, memSize) );
#else
		cuUtilsSafeCall( hipMalloc<float>(&d_steerFocusDelaySrc, memSize) );
		cuUtilsSafeCall( hipMemcpy(d_steerFocusDelaySrc, steerFocusDelaySrc, memSize, hipMemcpyHostToDevice) );
#endif

		float* d_srcTimeStamp = NULL;
#ifdef USE_CONST_MEMORY
		cuUtilsSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_srcTimeStamp), srcTimeStamp, memSize) );
#else
		cuUtilsSafeCall( hipMalloc<float>(&d_srcTimeStamp, memSize) );
		cuUtilsSafeCall( hipMemcpy(d_srcTimeStamp, srcTimeStamp, memSize, hipMemcpyHostToDevice) );
#endif

		unsigned int* d_srcPulseLength = NULL;
#ifdef USE_CONST_MEMORY
		cuUtilsSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_srcPulseLength), srcPulseLength, nSrc * sizeof(unsigned int)) );	
#else	
		cuUtilsSafeCall( hipMalloc<unsigned int>(&d_srcPulseLength, nSrc * sizeof(unsigned int)) );
		cuUtilsSafeCall( hipMemcpy(d_srcPulseLength, srcPulseLength, nSrc * sizeof(unsigned int), hipMemcpyHostToDevice) );
#endif

		hipComplex* d_res2;
		if (!resultOnGPU) {
			cuUtilsSafeCall( hipMalloc<hipComplex>(&d_res2, nObs * sizeof(hipComplex)) );
		}

		// Create block grid
		dim3 block(HOS_BLOCK_SIZE,1,1);
		dim3 grid((nObs - 1)/block.x + 1, 1);

		// launch kernel
		HuygensKernel<<<grid,block>>>(
			(resultOnGPU? d_res : d_res2), 
			nObs, d_coordObs, nSrc, d_coordSrc,
			d_fSrc, d_apodSrc, d_steerFocusDelaySrc, d_srcTimeStamp, d_srcPulseLength,
			timestampObs, refTime, c0);

		// check for errors
		hipDeviceSynchronize();

		cuUtilsSafeCall( hipGetLastError() );

		if (!resultOnGPU) {
			cuUtilsSafeCall( hipMemcpy(d_res, d_res2, nObs * sizeof(hipComplex), hipMemcpyDeviceToHost) );
			cuUtilsSafeCall( hipFree(d_res2) );	
		}

		// free device memory
		cuUtilsSafeCall( hipFree(d_apodSrc) );
		cuUtilsSafeCall( hipFree(d_coordObs) );
		cuUtilsSafeCall( hipFree(d_coordSrc) );
		cuUtilsSafeCall( hipFree(d_fSrc) );
		cuUtilsSafeCall( hipFree(d_steerFocusDelaySrc) );
		cuUtilsSafeCall( hipFree(d_srcTimeStamp) );
		cuUtilsSafeCall( hipFree(d_srcPulseLength) );

#ifdef USE_CONST_MEMORY
	}
#endif

	//free((void *)apodSrc);
	//free((void *)coordObs); // this one is now cleaned up by the observation object
	//free((void *)coordSrc);
	//free((void *)fSrc);
	//free((void *)steerFocusDelaySrc);
	//free((void *)srcTimeStamp);
	//free((void *)srcPulseLength);
}